#include<omp.h>
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<float.h>
#include <time.h>
#include <sys/time.h>
#define I 3
#define N 1024
#define M 1024
#define blockx 16
#define blocky 32
#define Thread_num 512
#define S 0
#define E 0
double cpuSecond() 
{
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}
void generate_matrix(double* matrix,int m,int n)
{
    for(int i=1;i<m;i++){
        for(int j=1;j<n;j++){
            matrix[i * n + j] =  (((double)rand())/RAND_MAX)*200;
        }
    }
    for(int i=0;i<m;i++){
        matrix[i*n] =  (((double)rand())/RAND_MAX)*200+200*(rand()%3)+200;
    }
    for(int i=0;i<n;i++)
    {
        matrix[i]=-(((double)rand())/RAND_MAX)*200;
    }
}

void read_matrix(double* matrix,int m,int n)
{
   
    FILE *fpWriteA=fopen("A.txt","r");
    FILE *fpWriteb=fopen("b.txt","r");
    FILE *fpWritec=fopen("c.txt","r");
    for(int i=1;i<m;i++)
    {
        for(int j=1;j<n;j++)
        {
            fscanf(fpWriteA," %lf ",matrix+i*N+j);
        }
        
    }
    for(int j=1;j<n;j++)
    {
        fscanf(fpWritec,"%lf",matrix+j);
    }
    for(int i=1;i<m;i++)
    {
        fscanf(fpWriteb,"%lf",matrix+i*n);
    }
    fclose (fpWriteA);
    fclose (fpWriteb);
    fclose (fpWritec);
}
void write_matrix(double* matrix,int m,int n)
{
    FILE *fpWriteA=fopen("A.txt","w");
    FILE *fpWriteb=fopen("b.txt","w");
    FILE *fpWritec=fopen("c.txt","w");
    for(int i=1;i<m;i++)
    {
        for(int j=1;j<n;j++)
        {
            fprintf(fpWriteA," %lf ",matrix[i*N+j]);
        }
        fprintf(fpWriteA,"\n");
    }
    for(int j=1;j<n;j++)
    {
        fprintf(fpWritec,"%lf\n",matrix[j]);
    }
    for(int i=1;i<m;i++)
    {
        fprintf(fpWriteb,"%lf\n",matrix[i*n]);
    }
    fclose (fpWriteA);
    fclose (fpWriteb);
    fclose (fpWritec);
}
int Find_min(double* array,int length)
{
    double min=DBL_MAX;
    int min_index=-1;
    for(int i=0;i<length;i++)
    {
    
    if(array[i]<min)
    { 
        min_index=i;
        min=array[i];

    }
    }
    return min_index;
}
__global__ void kernel1(double* theta,double* Columnk,int k,double* SimplexTableauPart,int size)
{
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
    if(idx<N)
    {
    double w=SimplexTableauPart[idx*N+k];
    Columnk[idx]=w;
    theta[idx]=((w>0) ? SimplexTableauPart[idx*N]/w : DBL_MAX);
    }
}
__global__ void kernel1_0(double* theta,double* Columnk,int k,double* SimplexTableauPart,int size)
{
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
    if(idx>0&&idx<N)
    {
    double w=SimplexTableauPart[idx*N+k];
    Columnk[idx]=w;
    theta[idx]=((w>0) ? SimplexTableauPart[idx*N]/w : DBL_MAX);
    }
    else
    {
        double w=SimplexTableauPart[idx*N+k];
        Columnk[idx]=w;
        theta[idx]=DBL_MAX;  
    }
}
__global__ void kernel2(double wp,int r,double *Columnk,double* Liner,double* SimplexTableauPart)
{
    int idx = blockDim.x*blockIdx.x+threadIdx.x;
    if(idx==0) Columnk[r]=-1;
    if(idx<N)
      Liner[idx] = SimplexTableauPart[r*N+idx]/wp;
}
__global__ void Kernel3(int size,double* Columnk,double* Liner,double* SimplexTableauPart)
{
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
    int idy=blockDim.y*blockIdx.y+threadIdx.y;
    if(idy<size&&idx<N)
    {
    double s = SimplexTableauPart[idy*N+idx];
    __shared__ double w[blocky];
    if(threadIdx.x==0)
      w[threadIdx.y] = Columnk[idy];
    __syncthreads();
    SimplexTableauPart[idy*N+idx]=s-w[threadIdx.y]*Liner[idx];
   }
}
__global__ void Kernel3_0(int size,double* Columnk, double* Liner,double* SimplexTableauPart)
{   
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
    int idy=blockDim.y*blockIdx.y+threadIdx.y;
    if(idx!=0||idy!=0)
    {
      if(idy<size&&idx<N)
       {
          double s = SimplexTableauPart[idy*N+idx];
          __shared__ double w[blocky];
          if(threadIdx.x==0||(blockIdx.x==0&&blockIdx.y==0&&threadIdx.x==1))
               w[threadIdx.y] = Columnk[idy];
          __syncthreads();
          SimplexTableauPart[idy*N+idx]=s-w[threadIdx.y]*Liner[idx];
       }
   }
}
__global__ void Kernel4(int size,int k,double wp,double* Columnk,double* SimplexTableauPart)
{
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
    if(idx<size)
    SimplexTableauPart[idx*N+k]=-Columnk[idx]/wp;
}
int main()
{   
    srand(time(NULL));
    bool label=true;
    int k,r,size,nsize,m0,n0,id;
    double min,wp;
    int* index,*index1,*Min;
    double* Sharedrow,*SimplexTableau,*SimplexTableauPart,*Columnk,*Liner,*LinerCPU,*theta;
    double thread_start_time;
    double thread_run_time;
    double time;
    m0=(M+I-1)/I;
    n0=(N+I-1)/I;
    Min=(int*)malloc(sizeof(int)*I);
    index=(int*)malloc(sizeof(int)*(M-1));
    index1=(int*)malloc(sizeof(int)*(N-1));
    Sharedrow=(double*)malloc(sizeof(double)*I*(n0>m0 ? n0 : m0));
    SimplexTableau=(double*)malloc(sizeof(double)*M*N);
    LinerCPU=(double*)malloc(sizeof(double)*N);
    //generate_matrix(SimplexTableau,M,N);
    read_matrix(SimplexTableau,M,N);
    //write_matrix(SimplexTableau,M,N);
    SimplexTableau[0]=DBL_MAX;
    if(S==1){
    printf("\n start \n");
    for(int i=0;i<M;i++){
        for(int j=0;j<N;j++){
        if(i==0&&j==0){
            printf(" CCC ");
            continue;
        }
        printf(" %.2f ",SimplexTableau[i*N+j]);
        }
        printf("\n");
    }
   }
    for(int i=0;i<M-1;i++)
    {
        index[i]=i+N;
    }
    for(int i=0;i<N-1;i++)
    {
        index1[i]=i+1;
    }
    #pragma omp parallel num_threads(I) private(theta,SimplexTableauPart,size,nsize,Columnk,Liner) shared(min,index,index1,Sharedrow,k,Min,m0,n0,id,LinerCPU,wp)
    {   
        
        int tid=omp_get_thread_num();
        hipSetDevice(tid);
        if(tid==(I-1))
          {
              size=M-m0*(I-1);
              nsize=N-n0*(I-1);
          }
        else
          {          
              size=m0;
              nsize=n0;
          }
        hipMalloc((void**)&Columnk,sizeof(double)*size);
        hipMalloc((void**)&theta,sizeof(double)*size);
        hipMalloc((void**)&SimplexTableauPart,sizeof(double)*size*N);
        hipMalloc((void**)&Liner,sizeof(double)*N);
        hipMemcpy(SimplexTableauPart,SimplexTableau+N*m0*tid,sizeof(double)*size*N,hipMemcpyHostToDevice);
        thread_start_time=cpuSecond();
        do
        {
        if(tid==0)
            hipMemcpy(Sharedrow,SimplexTableauPart,sizeof(double)*N,hipMemcpyDeviceToHost);
        {
        #pragma omp barrier
        }
        Min[tid]=Find_min(Sharedrow+tid*n0,nsize)+tid*n0;
        {
        #pragma omp barrier
        }
        if(tid==0)
        {
         k=Min[0];
         min=Sharedrow[Min[0]];
         for(int i=1;i<I;i++)
           {   
            if(Sharedrow[Min[i]]<min)
            {   
                k=Min[i];
                min=Sharedrow[k];
            }
           }
       }
      {
       #pragma omp barrier
       }
       if(min>=0) break;
       if(tid==0)
           kernel1_0<<<(size+Thread_num-1)/Thread_num,Thread_num>>>(theta,Columnk,k,SimplexTableauPart,size);
       else
           kernel1<<<(size+Thread_num-1)/Thread_num,Thread_num>>>(theta,Columnk,k,SimplexTableauPart,size);
       hipMemcpy(Sharedrow+(tid)*m0,theta,sizeof(double)*size,hipMemcpyDeviceToHost);
       {
        #pragma omp barrier
       }
       Min[tid]=Find_min(Sharedrow+(tid)*m0,size);
       Min[tid]=((Min[tid]<0)?-1:(Min[tid]+tid*m0));
       {
       #pragma omp barrier
       }
       if(tid==0)
        {
         r=-1;
         double min=DBL_MAX;
         for(int i=0;i<I;i++)  
            if(Min[i]>-1&&Sharedrow[Min[i]]<min)
              {
                 
                 r=Min[i];
                 id=i;
                 min=Sharedrow[r];
              }
        }
       {
       #pragma omp barrier
       }
       
       if(r==-1) 
       {   
           label=false;
           break;
       }
       if(tid==id)
       {   
           int tem=index[r-1];
           index[r-1]=index1[k-1];
           index1[k-1]=tem;
           hipMemcpy(&wp,SimplexTableauPart+(r-tid*m0)*N+k,sizeof(double),hipMemcpyDeviceToHost);
           kernel2<<<(N+Thread_num-1)/Thread_num,Thread_num>>>(wp,r-tid*m0,Columnk,Liner,SimplexTableauPart) ;
           hipMemcpy(LinerCPU,Liner,sizeof(double)*N,hipMemcpyDeviceToHost);
           hipMemset(SimplexTableauPart+(r-tid*m0)*N,0.0,N*sizeof(double));
        }
        {
        #pragma omp barrier
        }
       hipMemcpy(Liner,LinerCPU,sizeof(double)*N,hipMemcpyHostToDevice);
       dim3 block_size(blockx,blocky);
       dim3 grid_size((N+blockx-1)/blockx,(size+blocky-1)/blocky);
       if(tid==0)
           Kernel3_0<<<grid_size,block_size>>>(size,Columnk,Liner,SimplexTableauPart);
       else
           Kernel3<<<grid_size,block_size>>>(size,Columnk,Liner,SimplexTableauPart);
      
       Kernel4<<<(size+Thread_num-1)/Thread_num,Thread_num>>>(size,k,wp,Columnk,SimplexTableauPart);
       hipDeviceSynchronize();
        {
            #pragma omp barrier
        }
           
            
       }while(1);
       thread_run_time=cpuSecond()-thread_start_time;
       printf("thread %d run time is %f \n",tid,thread_run_time);
       //cudaMemcpy(SimplexTableau+N*m0*tid,SimplexTableauPart,sizeof(double)*size*N,cudaMemcpyDeviceToHost);
       hipFree(SimplexTableauPart);        
    }
    if(E==1)
    {
    printf("\n end \n ");
    for(int i=0;i<M;i++){
        for(int j=0;j<N;j++){
        if(i==0&&j==0){
            printf(" CCC ");
            continue;
        }
        printf(" %.2f ",SimplexTableau[i*N+j]);
        }
        printf("\n");
    }
   }
    double x_result[M-1];
    for(int i=0;i<M-1;i++)
    {
          x_result[i]=0;
    }
    for(int i=0;i<M-1;i++)
    {   
        if(index[i]<=M-1)
          x_result[index[i]-1]=SimplexTableau[(i+1)*N];
    }
    
    if(label){
       printf("\n true \n");
       FILE *fpWriter=fopen("r.txt","w");
       for(int i=0;i<M-1;i++){
            //printf(" the  x_%d is %f \n",i+1,x_result[i]);
            fprintf(fpWriter,"%lf\n",x_result[i]);
        }
        

    }
    else
    {
        printf("\n false \n");
    }
    free(SimplexTableau);
    return 0;
}