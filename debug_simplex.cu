#include<omp.h>
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<float.h>
#define I 2
#define N 5
#define M 5
#define blockx 2
#define blocky 2
#define Thread_num 2
#define J 1
#define K2 0
#define K3 0
#define BK3 0
void generate_matrix(double* matrix,int m,int n)
{
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            matrix[i * n + j] =  ((double)((rand()%10)+1));
        }
    }
    for(int i=0;i<m;i++){
        matrix[i*n] =  (double)((rand()%10)+1);
    }
    for(int i=0;i<n;i++)
    {
        matrix[i]=-matrix[i];
    }
}
void read_matrix(double* matrix,int m,int n)
{
    FILE *fpRead=fopen("data.txt","r");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            fscanf(fpRead,"%lf",matrix+i*N+j);
        }
    }

}
int Find_min(double* array,int length)
{
    double min=DBL_MAX;
    int min_index=-1;
    for(int i=0;i<length;i++)
    {
    
    if(array[i]<min)
    { 
        min_index=i;
        min=array[i];

    }
    }
    return min_index;
}
__global__ void kernel1(double* theta,double* Columnk,int k,double* SimplexTableauPart,int size)
{
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
    if(idx<N)
    {
    double w=SimplexTableauPart[idx*N+k];
    Columnk[idx]=w;
    theta[idx]=((w>0) ? SimplexTableauPart[idx*N]/w : DBL_MAX);
    }
}
__global__ void kernel1_0(double* theta,double* Columnk,int k,double* SimplexTableauPart,int size)
{
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
    if(idx>0&&idx<N)
    {
    double w=SimplexTableauPart[idx*N+k];
    Columnk[idx]=w;
    theta[idx]=((w>0) ? SimplexTableauPart[idx*N]/w : DBL_MAX);
    }
    else
    {
        double w=SimplexTableauPart[idx*N+k];
        Columnk[idx]=w;
        theta[idx]=DBL_MAX;  
    }
}
__global__ void kernel2(double wp,int r,double *Columnk,double* Liner,double* SimplexTableauPart)
{
    int idx = blockDim.x*blockIdx.x+threadIdx.x;
    if(idx==0) Columnk[r]=-1;
    if(idx<N)
      Liner[idx] = SimplexTableauPart[r*N+idx]/wp;
}
__global__ void Kernel3(int size,double* Columnk,double* Liner,double* SimplexTableauPart)
{
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
    int idy=blockDim.y*blockIdx.y+threadIdx.y;
    if(idy<size&&idx<N)
    {
    double s = SimplexTableauPart[idy*N+idx];
    __shared__ double w[blocky];
    if(threadIdx.x==0)
      w[threadIdx.y] = Columnk[idy];
    __syncthreads();
    SimplexTableauPart[idy*N+idx]=s-w[threadIdx.y]*Liner[idx];
   }
}
__global__ void Kernel3_0(int size,double* Columnk, double* Liner,double* SimplexTableauPart)
{   
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
    int idy=blockDim.y*blockIdx.y+threadIdx.y;
    if(idx!=0||idy!=0)
    {
      if(idy<size&&idx<N)
       {
          double s = SimplexTableauPart[idy*N+idx];
          __shared__ double w[blocky];
          if(threadIdx.x==0||(blockIdx.x==0&&blockIdx.y==0&&threadIdx.x==1))
               w[threadIdx.y] = Columnk[idy];
          __syncthreads();
          SimplexTableauPart[idy*N+idx]=s-w[threadIdx.y]*Liner[idx];
       }
   }
}
__global__ void Kernel4(int size,int k,double wp,double* Columnk,double* SimplexTableauPart)
{
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
    if(idx<size)
    SimplexTableauPart[idx*N+k]=-Columnk[idx]/wp;
}
int main()
{   
    bool label=true;
    int k,r,size,nsize,m0,n0,id;
    double min,wp;
    int* index,*index1,*Min;
    double* Sharedrow,*SimplexTableau,*SimplexTableauPart,*Columnk,*Liner,*LinerCPU,*theta;
    m0=(M+I-1)/I;
    n0=(N+I-1)/I;
    Min=(int*)malloc(sizeof(int)*I);
    index=(int*)malloc(sizeof(int)*(M-1));
    index1=(int*)malloc(sizeof(int)*(N-1));
    Sharedrow=(double*)malloc(sizeof(double)*I*(n0>m0 ? n0 : m0));
    SimplexTableau=(double*)malloc(sizeof(double)*M*N);
    LinerCPU=(double*)malloc(sizeof(double)*N);
    generate_matrix(SimplexTableau,M,N);
    //read_matrix(SimplexTableau,M,N);
    SimplexTableau[0]=DBL_MAX;
    for(int i=0;i<M-1;i++)
    {
        index[i]=i+N;
    }
    for(int i=0;i<N-1;i++)
    {
        index1[i]=i+1;
    }
    printf("start \n ");
    for(int i=0;i<M;i++){
        for(int j=0;j<N;j++){
        if(i==0&&j==0){
            printf(" CCC ");
            continue;
        }
        printf(" %.2f ",SimplexTableau[i*N+j]);
        }
        printf("\n");
    }
    
    #pragma omp parallel num_threads(I) private(theta,SimplexTableauPart,size,nsize,Columnk,Liner) shared(min,index,index1,Sharedrow,k,Min,m0,n0,id,LinerCPU,wp)
    {   
        
        int tid=omp_get_thread_num();
        hipSetDevice(tid);
        if(tid==(I-1))
          {
              size=M-m0*(I-1);
              nsize=N-n0*(I-1);
          }
        else
          {          
              size=m0;
              nsize=n0;
          }
        hipMalloc((void**)&Columnk,sizeof(double)*size);
        hipMalloc((void**)&theta,sizeof(double)*size);
        hipMalloc((void**)&SimplexTableauPart,sizeof(double)*size*N);
        hipMalloc((void**)&Liner,sizeof(double)*N);
        hipMemcpy(SimplexTableauPart,SimplexTableau+N*m0*tid,sizeof(double)*size*N,hipMemcpyHostToDevice);
        do
        {
        if(tid==0)
            hipMemcpy(Sharedrow,SimplexTableauPart,sizeof(double)*N,hipMemcpyDeviceToHost);
        {
        #pragma omp barrier
        }
        Min[tid]=Find_min(Sharedrow+tid*n0,nsize)+tid*n0;
        {
        #pragma omp barrier
        }
        if(tid==0)
        {
         k=Min[0];
         min=Sharedrow[Min[0]];
         for(int i=1;i<I;i++)
           {   
            if(Sharedrow[Min[i]]<min)
            {   
                k=Min[i];
                min=Sharedrow[k];
            }
           }
        printf(" \n k is %d with value %f\n ",k,min);
      }
      {
       #pragma omp barrier
       }
       if(min>=0&&J==1) break;
       if(tid==0)
           kernel1_0<<<(size+Thread_num-1)/Thread_num,Thread_num>>>(theta,Columnk,k,SimplexTableauPart,size);
       else
           kernel1<<<(size+Thread_num-1)/Thread_num,Thread_num>>>(theta,Columnk,k,SimplexTableauPart,size);
       hipMemcpy(Sharedrow+(tid)*m0,theta,sizeof(double)*size,hipMemcpyDeviceToHost);
       {
        #pragma omp barrier
       }
       Min[tid]=Find_min(Sharedrow+(tid)*m0,size);
       Min[tid]=((Min[tid]<0)?-1:(Min[tid]+tid*m0));
       {
       #pragma omp barrier
       }
       if(tid==0)
        {
         r=-1;
         double min=DBL_MAX;
         for(int i=0;i<I;i++)  
            if(Min[i]>-1&&Sharedrow[Min[i]]<min)
              {
                 
                 r=Min[i];
                 id=i;
                 min=Sharedrow[r];
              }
         if(r!=-1)
           printf("\n r is %d with value of %f \n",r,min);
         else
           printf("\n r is -1 !!!\n");
        }
       {
       #pragma omp barrier
       }
       
       if(r==-1&&J==1) 
       {   
           label=false;
           break;
       }
       if(tid==id)
       {   
           int tem=index[r-1];
           index[r-1]=index1[k-1];
           index1[k-1]=tem;
           wp=SimplexTableau[r*N+k];
           kernel2<<<(N+Thread_num-1)/Thread_num,Thread_num>>>(wp,r-tid*m0,Columnk,Liner,SimplexTableauPart) ;
           hipMemcpy(LinerCPU,Liner,sizeof(double)*N,hipMemcpyDeviceToHost);
           hipMemset(SimplexTableauPart+(r-tid*m0)*N,0.0,N*sizeof(double));
        }
        {
        #pragma omp barrier
        }
       hipMemcpy(Liner,LinerCPU,sizeof(double)*N,hipMemcpyHostToDevice);
       dim3 block_size(blockx,blocky);
       dim3 grid_size((N+blockx-1)/blockx,(size+blocky-1)/blocky);
       if(tid==0)
           Kernel3_0<<<grid_size,block_size>>>(size,Columnk,Liner,SimplexTableauPart);
       else
           Kernel3<<<grid_size,block_size>>>(size,Columnk,Liner,SimplexTableauPart);
       Kernel4<<<(size+Thread_num-1)/Thread_num,Thread_num>>>(size,k,wp,Columnk,SimplexTableauPart);
       hipDeviceSynchronize();
       
       hipMemcpy(SimplexTableau+N*m0*tid,SimplexTableauPart,sizeof(double)*size*1,hipMemcpyDeviceToHost);
       {
            #pragma omp barrier
        }
           
       }while(J==1);
       hipMemcpy(SimplexTableau+N*m0*tid,SimplexTableauPart,sizeof(double)*size*N,hipMemcpyDeviceToHost);
       hipFree(SimplexTableauPart);        
    }
    if(label){
        printf("\n true \n");
        for(int i=0;i<M-1;i++){
            printf("the index i is %d \n",index[i]);
        }
    }
    else
    {
        printf("\n false \n");
    }
    
    
    printf("\n end \n ");
    for(int i=0;i<M;i++){
        for(int j=0;j<N;j++){
        if(i==0&&j==0){
            printf(" CCC ");
            continue;
        }
        printf(" %.2f ",SimplexTableau[i*N+j]);
        }
        printf("\n");
    }
    
    free(SimplexTableau);
    return 0;
}